#include "hip/hip_runtime.h"
#include <iostream>
#include <thread>
#include <igl/opengl/glfw/Viewer.h>
#include <hip/hip_runtime.h>

#include <init_particles.cuh>
#include <mem_util.cuh>
#include <simulation_step.cuh>

#include <time.h>


Eigen::MatrixXd positions;
float3* velocity;
float* cpu_device_buf;
float3* positions_device;
float3* positions_star_device;

int* result;
int* grid_index;
int* particle_index;
int* cell_start;
int* cell_end;
float* lambdas;
float3* delta_positions;

double particle_init_step = 0.1;
igl::opengl::glfw::Viewer viewer;

float3* sim_space_bot_left;
float3* sim_space_top_right;


//constants
float3* gravity_m;

//simulation time and time step
float t = 0; //simulation time 
float dt = 0.01; //time step
float cube_s = 0.4;
float h = cube_s;
float mass = 0.8;
float pho0 = 8000.0;
float epsilon = 1000;
float num_iteration = 3;

const Eigen::RowVector3d particle_color(0.333, 0.647, 0.905);
const int xid = viewer.selected_data_index;

//simulation loop
bool simulating = false;

bool simulation_callback() {

    while (simulating) {
    
        // simulation_step(positions, cpu_device_buf, positions_device, positions_star_device,
        // velocity, gravity_m, sim_space_bot_left, sim_space_top_right, result,
        // grid_index, particle_index, cell_start, cell_end, lambdas, delta_positions,
        // cube_s, dt, h, mass, pho0, epsilon, num_iteration);
    }
    return true;
}

bool draw_callback(igl::opengl::glfw::Viewer &viewer) {

    simulation_step(positions, cpu_device_buf, positions_device, positions_star_device,
        velocity, gravity_m, sim_space_bot_left, sim_space_top_right, result,
        grid_index, particle_index, cell_start, cell_end, lambdas, delta_positions,
        cube_s, dt, h, mass, pho0, epsilon, num_iteration, simulating);


    viewer.data_list[xid].set_points(positions, particle_color);
    return false;
}


int main(int argc, char **argv) {

    std::cout<<"Start Project\n";
    int num_cell = 8000;

    //setup libigl viewer and activate 

    viewer.core().background_color.setConstant(1.0);

    hipError_t status;
    if ((status = hipMalloc(&sim_space_bot_left, sizeof(float3))) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    if ((status = hipMalloc(&sim_space_top_right, sizeof(float3))) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    if ((status = hipMalloc(&gravity_m, sizeof(float3))) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    cpu_device_buf = (float*)malloc(sizeof(float)*3);
    Eigen::MatrixXd tmp(1, 3);
    tmp << -1.0, -1.0, -1.0;
    to_gpu(tmp, cpu_device_buf, sim_space_bot_left);
    tmp << 7.0, 7.0, 7.0;
    to_gpu(tmp, cpu_device_buf, sim_space_top_right);
    tmp << 0.0, -9.8, 0.0;
    to_gpu(tmp, cpu_device_buf, gravity_m);
    free(cpu_device_buf);

    Eigen::Vector3d particle_init_bot_left;
    particle_init_bot_left << 1.0, 1.0, 1.0;

    init_particles(positions, particle_init_bot_left, particle_init_step, 
    20, 30, 20);
    cpu_device_buf = (float*)malloc(sizeof(float)*3*positions.rows());
    
    if ((status = hipMalloc(&positions_device, sizeof(float3)*positions.rows())) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    if ((status = hipMalloc(&positions_star_device, sizeof(float3)*positions.rows())) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    if ((status = hipMalloc(&velocity, sizeof(float3)*positions.rows())) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    if ((status = hipMalloc(&result, sizeof(int)*positions.rows())) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    if ((status = hipMalloc(&grid_index, sizeof(int)*positions.rows())) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    if ((status = hipMalloc(&particle_index, sizeof(int)*positions.rows())) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    if ((status = hipMalloc(&cell_start, sizeof(int)*num_cell)) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    if ((status = hipMalloc(&cell_end, sizeof(int)*num_cell)) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    if ((status = hipMalloc(&lambdas, sizeof(float)*positions.rows())) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }
    if ((status = hipMalloc(&delta_positions, sizeof(float3)*positions.rows())) != hipSuccess) {
        std::cout << "ERROR hipMalloc" << hipGetErrorName(status) << std::endl;
    }

    to_gpu(positions, cpu_device_buf, positions_device);
    // positions.setZero();
    // to_cpu(positions_device, cpu_device_buf, positions);

    viewer.data_list[xid].set_points(positions, particle_color);
    viewer.data_list[xid].point_size = 8;

    // Eigen::Vector3d g_v;
    // g_v << 0.0, -9.8, 0.0;

    // std::thread simulation_thread(simulation_callback);
    // simulation_thread.detach();

    viewer.callback_post_draw = &draw_callback;

    viewer.callback_key_pressed =
			[&](igl::opengl::glfw::Viewer&, unsigned char key, int)->bool
		{
			switch (key)
			{
			case 'A':
			case 'a':
				//with ghost pressure
				simulating = !simulating;
                // std::cout << positions << std::endl;
				break;
			default:
				return false;
			}
			return true;
		};

    viewer.launch();

    return 0;
}
