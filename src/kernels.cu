#include "hip/hip_runtime.h"
#include <kernels.cuh>

__device__ float poly6(float r, float h) {
    if (r >= 0 && r <= h) {
        return (315.0 * powf(h*h-r*r, 3.0)) / (64.0 * M_PI * powf(h, 9.0));
    }
    return 0.0;
}

__device__ void spiky_grad(float3 *r, float h, float3 *grad) {
    float norm = norm3df(r->x, r->y, r->z);
    float3 normalized = make_float3(r->x / norm, r->y / norm, r->z / norm);
    if (norm > 0 && norm <= h) {
        float coeff = -(45.0 * powf(norm - h, 2.0)) / (M_PI * powf(h, 6.0));
        grad->x = normalized.x * coeff; 
        grad->y = normalized.y * coeff; 
        grad->z = normalized.z * coeff; 
    } else {
        grad->x = 0.0;
        grad->y = 0.0;
        grad->z = 0.0;
    }
}