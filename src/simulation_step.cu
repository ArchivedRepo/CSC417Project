#include "hip/hip_runtime.h"
#include <simulation_step.cuh>
#include <advect.cuh>
#include <mem_util.cuh>
#include <build_grid.cuh>
#include <compute_lambda.cuh>
#include <compute_delta_position.cuh>
#include <update_position.cuh>
#include <update_velocity.cuh>
#include <viscosity_confinement.cuh>
#include <iostream>
#include <time.h>

void simulation_step(
    Eigen::MatrixXd &positions,
    float* cpu_device_buf,
    float3* device_positions,
    float3* device_positions_star,
    float3* velocity,
    float3* gravity,
    float3* sim_space_bot_left,
    float3* sim_space_top_right,
    int* result,
    int* grid_index,
    int* particle_index,
    int* cell_start,
    int* cell_end,
    float* lambdas,
    float3* delta_positions,
    float cube_s,
    float dt,
    float h,
    float mass,
    float pho0, 
    float epsilon,
    float num_iteration,
    bool simulation
) { 
    if (simulation){
        clock_t t = clock();

        int N = positions.rows();

        dim3 grid_dim(ceil((double) N /128), 1, 1);
        dim3 thread_block(128, 1, 1);
        advect<<<grid_dim, thread_block>>>(device_positions, device_positions_star,
        velocity, gravity, sim_space_bot_left, sim_space_top_right, dt, N);
        hipDeviceSynchronize();

        build_grid(device_positions_star, result, cube_s, sim_space_bot_left,
        sim_space_top_right, grid_index, particle_index, cell_start, cell_end, N);
        hipDeviceSynchronize();

        for (int iter=0; iter < num_iteration; iter++) {
            compute_lambda<<<grid_dim, thread_block>>>(device_positions_star, pho0, mass, epsilon, h,
            lambdas, cell_start, cell_end, grid_index, particle_index,
            sim_space_bot_left, sim_space_top_right, cube_s, N);
            hipDeviceSynchronize();

            compute_delta_position<<<grid_dim, thread_block>>>(device_positions_star, pho0, h, lambdas, delta_positions,
            cell_start, cell_end, grid_index, particle_index, 
            sim_space_bot_left, sim_space_top_right, cube_s, N);

            update_positions<<<grid_dim, thread_block>>>(device_positions_star, delta_positions, N);
        }
        update_velocity<<<grid_dim, thread_block>>>(device_positions, device_positions_star, velocity, dt, N);
        
        viscosity_confinement<<<grid_dim, thread_block>>>(device_positions_star, velocity, h, cell_start, cell_end,
        grid_index, particle_index, sim_space_bot_left, sim_space_top_right, cube_s, N);
        

        hipError_t status;
        if ((status = hipMemcpy(device_positions, device_positions_star, N*sizeof(float)*3,hipMemcpyDeviceToDevice))!= hipSuccess) {
            std::cout << "ERROR memcpy: " << hipGetErrorName(status) << std::endl;
        }
        to_cpu(device_positions, cpu_device_buf, positions);

        hipDeviceSynchronize();

        t = clock() - t;
        double diff = (double)t / CLOCKS_PER_SEC;
        std::cout << diff << std::endl; 
    }
}