#include "hip/hip_runtime.h"
#include <mem_util.cuh>
#include <iostream>

void to_gpu(
    Eigen::MatrixXd &src,
    float* buf,
    float3* dest
) {
    for (int i =0; i< src.rows(); i++) {
        buf[i*3+0] = src(i, 0);
        buf[i*3+1] = src(i, 1);
        buf[i*3+2] = src(i, 2);
    }
    hipError_t status;
    if ((status = hipMemcpy(dest, buf, src.rows()*3*sizeof(float), hipMemcpyHostToDevice)) != hipSuccess) {
        std::cout << "to_gpu: " << hipGetErrorName(status) <<std::endl;
    }
}

void to_cpu(
    float3* src,
    float* buf,
    Eigen::MatrixXd &dest
) {
    hipError_t status;
    if ((status = hipMemcpy(buf, src, dest.rows()*3*sizeof(float), hipMemcpyDeviceToHost)) != hipSuccess) {
        std::cout << "to_cpu: " << hipGetErrorName(status) <<std::endl;
    }
    int pos = 0;
    for (int i =0; i < dest.rows(); i++) {
        dest.row(i) << buf[pos], buf[pos+1], buf[pos+2];
        pos += 3;
    }
}